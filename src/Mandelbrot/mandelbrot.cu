﻿#include "cudaTimer.h"
#include "mandelbrot.h"
#include <GLAD/glad.h>
#include <GLFW/glfw3.h>

Mandelbrot::Mandelbrot(int width, int height, unsigned short int maxIterations)
    : width_(width), height_(height), maxIterations_(maxIterations) {
    this->h_data_ = new unsigned short int[width_ * height_];
    for (int i = 0; i < width_ * height_; i++)
        this->h_data_[i] = 0;
    CUDA_CHECK(hipMalloc((void **)&this->d_data_, width_ * height_ * sizeof(unsigned short int)));
}

Mandelbrot::~Mandelbrot() {
    delete[] this->h_data_;
    CUDA_CHECK(hipFree(this->d_data_));
}
