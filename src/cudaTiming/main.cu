﻿#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                                     \
    {                                                                                        \
        do {                                                                                 \
            const hipError_t error = call;                                                  \
            if (error != hipSuccess) {                                                      \
                fprintf(stderr, "CUDA Error: %s:%d\n", __FILE__, __LINE__);                  \
                fprintf(stderr, "code: %d\nreason: %s\n", error, hipGetErrorString(error)); \
                exit(EXIT_FAILURE);                                                          \
            }                                                                                \
        } while (0);                                                                         \
    }

int main(int argc, char const *argv[]) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    CUDA_CHECK(hipEventSynchronize(start));
    // 此处不能使用 CUDA_CHECK，因为返回值可能是 cudaErrorNotReady
    // cudaEventQuery(start);

    // 计时

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Elapsed time: %g ms\n", milliseconds);

    // 销毁事件
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
