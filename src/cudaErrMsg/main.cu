﻿#include "cudaErrMsg.h"

int main(int argc, char const *argv[]) {
    double *h_data = (double *)malloc(1024 * sizeof(double));
    double *d_data = nullptr;
    // 检查 CUDA 运行时 API 调用是否成功
    CUDA_CHECK(hipMalloc((void **)&d_data, 1024 * sizeof(double)));
    // 故意写错 hipMemcpyKind 以触发错误
    CUDA_CHECK(hipMemcpy(d_data, h_data, 1024 * sizeof(double), hipMemcpyDeviceToHost));

    // 运行一些 CUDA 内核（假设有一个内核函数）
    // kernel<<<1, 1>>>(d_data); // 假设有一个
    CUDA_CHECK(hipGetLastError());      // 检查内核执行是否有错误
    CUDA_CHECK(hipDeviceSynchronize()); // 确保所有 CUDA 操作完成

    // 释放设备内存
    CUDA_CHECK(hipFree(d_data));
    // 释放主机内存
    free(h_data);
    return 0;
}
